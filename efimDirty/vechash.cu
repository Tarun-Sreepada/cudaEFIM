#pragma once
#include <hip/hip_runtime.h>

#include <vector>

struct vector_hash
{
    std::size_t operator()(std::vector<uint32_t> const &vec) const
    {
        std::size_t seed = vec.size();
        for (auto x : vec)
        {
            x = ((x >> 16) ^ x) * 0x45d9f3b;
            x = ((x >> 16) ^ x) * 0x45d9f3b;
            x = (x >> 16) ^ x;
            seed ^= x + 0x9e3779b9 + (seed << 6) + (seed >> 2);
        }
        return seed;
    }
};
